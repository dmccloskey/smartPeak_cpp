
#include <hip/hip_runtime.h>
/**TODO:  Add copyright*/

#if COMPILE_WITH_CUDA

#include <SmartPeak/ml/ModelInterpreterGpu.h>

using namespace SmartPeak;
using namespace std;

Model<float> makeModelToy1()
{
	/**
	* Directed Acyclic Graph Toy Network Model
	*/
	Node<float> i1, i2, h1, h2, o1, o2, b1, b2;
	Link l1, l2, l3, l4, lb1, lb2, l5, l6, l7, l8, lb3, lb4;
	Weight<float> w1, w2, w3, w4, wb1, wb2, w5, w6, w7, w8, wb3, wb4;
	Model<float> model_FC_Sum;

	// Toy network: 1 hidden layer, fully connected, DAG
	i1 = Node<float>("0", NodeType::input, NodeStatus::activated, std::shared_ptr<ActivationOp<float>>(new LinearOp<float>()), std::shared_ptr<ActivationOp<float>>(new LinearGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	i2 = Node<float>("1", NodeType::input, NodeStatus::activated, std::shared_ptr<ActivationOp<float>>(new LinearOp<float>()), std::shared_ptr<ActivationOp<float>>(new LinearGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	h1 = Node<float>("2", NodeType::hidden, NodeStatus::initialized, std::shared_ptr<ActivationOp<float>>(new ReLUOp<float>()), std::shared_ptr<ActivationOp<float>>(new ReLUGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	h2 = Node<float>("3", NodeType::hidden, NodeStatus::initialized, std::shared_ptr<ActivationOp<float>>(new ReLUOp<float>()), std::shared_ptr<ActivationOp<float>>(new ReLUGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	o1 = Node<float>("4", NodeType::output, NodeStatus::initialized, std::shared_ptr<ActivationOp<float>>(new ReLUOp<float>()), std::shared_ptr<ActivationOp<float>>(new ReLUGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	o2 = Node<float>("5", NodeType::output, NodeStatus::initialized, std::shared_ptr<ActivationOp<float>>(new ReLUOp<float>()), std::shared_ptr<ActivationOp<float>>(new ReLUGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	b1 = Node<float>("6", NodeType::bias, NodeStatus::activated, std::shared_ptr<ActivationOp<float>>(new LinearOp<float>()), std::shared_ptr<ActivationOp<float>>(new LinearGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	b2 = Node<float>("7", NodeType::bias, NodeStatus::activated, std::shared_ptr<ActivationOp<float>>(new LinearOp<float>()), std::shared_ptr<ActivationOp<float>>(new LinearGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));

	// weights  
	std::shared_ptr<WeightInitOp<float>> weight_init;
	std::shared_ptr<SolverOp<float>> solver;
	// weight_init.reset(new RandWeightInitOp(1.0)); // No random init for testing
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	w1 = Weight<float>("0", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	w2 = Weight<float>("1", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	w3 = Weight<float>("2", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	w4 = Weight<float>("3", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	wb1 = Weight<float>("4", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	wb2 = Weight<float>("5", weight_init, solver);
	// input layer + bias
	l1 = Link("0", "0", "2", "0");
	l2 = Link("1", "0", "3", "1");
	l3 = Link("2", "1", "2", "2");
	l4 = Link("3", "1", "3", "3");
	lb1 = Link("4", "6", "2", "4");
	lb2 = Link("5", "6", "3", "5");
	// weights
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	w5 = Weight<float>("6", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	w6 = Weight<float>("7", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	w7 = Weight<float>("8", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	w8 = Weight<float>("9", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	wb3 = Weight<float>("10", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	wb4 = Weight<float>("11", weight_init, solver);
	// hidden layer + bias
	l5 = Link("6", "2", "4", "6");
	l6 = Link("7", "2", "5", "7");
	l7 = Link("8", "3", "4", "8");
	l8 = Link("9", "3", "5", "9");
	lb3 = Link("10", "7", "4", "10");
	lb4 = Link("11", "7", "5", "11");
	model_FC_Sum.setId(1);
	model_FC_Sum.addNodes({ i1, i2, h1, h2, o1, o2, b1, b2 });
	model_FC_Sum.addWeights({ w1, w2, w3, w4, wb1, wb2, w5, w6, w7, w8, wb3, wb4 });
	model_FC_Sum.addLinks({ l1, l2, l3, l4, lb1, lb2, l5, l6, l7, l8, lb3, lb4 });
	return model_FC_Sum;
}

void test_allocateForwardPropogationLayerTensors()
{
	Model<float> model_allocateForwardPropogationLayerTensors = makeModelToy1();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 2;
	const bool train = true;

	// initialize nodes
	// NOTE: input and biases have been activated when the model was created

	// Check iteration one with no source/sink/weight tensors already allocated
	std::map<std::string, int> FP_operations_map;
	std::vector<OperationList<float>> FP_operations_list;
	model_interpreter.getNextInactiveLayer(model_allocateForwardPropogationLayerTensors, FP_operations_map, FP_operations_list);

	std::vector<std::string> sink_nodes_with_biases2;
	model_interpreter.getNextInactiveLayerBiases(model_allocateForwardPropogationLayerTensors, FP_operations_map, FP_operations_list, sink_nodes_with_biases2);

	std::vector<OperationList<float>> FP_operations_expanded;
	model_interpreter.expandForwardPropogationOperations(FP_operations_list, FP_operations_expanded);

	std::set<std::string> identified_sink_nodes;
	std::map<std::string, std::vector<int>> tensor_ops = model_interpreter.getTensorOperations(FP_operations_expanded, identified_sink_nodes);

	std::vector<int> source_layer_sizes, sink_layer_sizes;
	std::vector<std::vector<std::pair<int, int>>> weight_indices;
	std::vector<std::vector<float>> weight_values;
	std::vector<bool> make_source_tensors, make_sink_tensors, make_weight_tensors;
	model_interpreter.getForwardPropogationLayerTensorDimensions(FP_operations_expanded, tensor_ops, source_layer_sizes, sink_layer_sizes, weight_indices, weight_values, make_source_tensors, make_sink_tensors, make_weight_tensors);
	model_interpreter.allocateForwardPropogationLayerTensors(FP_operations_expanded, tensor_ops, source_layer_sizes, sink_layer_sizes, weight_indices, weight_values, make_source_tensors, make_sink_tensors, make_weight_tensors, batch_size, memory_size, train);

	// asserts are needed because boost deallocates the pointer memory after being called...
	assert(model_interpreter.getLayerTensor(0)->getBatchSize() == batch_size); // sinks
	assert(model_interpreter.getLayerTensor(0)->getMemorySize() == memory_size); // sinks
	assert(model_interpreter.getLayerTensor(0)->getLayerSize() == 2); // sinks
	assert(model_interpreter.getLayerTensor(1)->getBatchSize() == batch_size); // sources
	assert(model_interpreter.getLayerTensor(1)->getMemorySize() == memory_size); // sources
	assert(model_interpreter.getLayerTensor(1)->getLayerSize() == 3); // sources
	assert(model_interpreter.getWeightTensor(0)->getLayer1Size() == 3);
	assert(model_interpreter.getWeightTensor(0)->getLayer2Size() == 2);
	assert(model_interpreter.getWeightTensor(0)->getNSolverParams() == 3);
	assert(model_interpreter.getOperationSteps(0)[0].source_layer.time_step == 0);
	assert(model_interpreter.getOperationSteps(0)[0].source_layer.activation->getName() == "LinearTensorOp");
	assert(model_interpreter.getOperationSteps(0)[0].source_layer.activation_grad->getName() == "LinearGradTensorOp");
	assert(model_interpreter.getOperationSteps(0)[0].source_layer.integration->getName() == "SumTensorOp");
	assert(model_interpreter.getOperationSteps(0)[0].source_layer.integration_error->getName() == "SumErrorTensorOp");
	assert(model_interpreter.getOperationSteps(0)[0].source_layer.integration_weight_grad->getName() == "SumWeightGradTensorOp");
	assert(model_interpreter.getOperationSteps(0)[0].sink_layer.time_step == 0);
	assert(model_interpreter.getOperationSteps(0)[0].sink_layer.activation->getName() == "ReLUTensorOp");
	assert(model_interpreter.getOperationSteps(0)[0].sink_layer.activation_grad->getName() == "ReLUGradTensorOp");
	assert(model_interpreter.getOperationSteps(0)[0].sink_layer.integration->getName() == "SumTensorOp");
	assert(model_interpreter.getOperationSteps(0)[0].sink_layer.integration_error->getName() == "SumErrorTensorOp");
	assert(model_interpreter.getOperationSteps(0)[0].sink_layer.integration_weight_grad->getName() == "SumWeightGradTensorOp");
	assert(model_interpreter.getOperationSteps(0)[0].weight.solver->getName() == "SGDTensorOp");
}

void test_getForwardPropogationOperations()
{
	Model<float> model_getForwardPropogationOperations = makeModelToy1();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 1;
	const bool train = true;

	// initialize nodes
	// NOTE: input and biases have been activated when the model was created

	model_interpreter.getForwardPropogationOperations(model_getForwardPropogationOperations, batch_size, memory_size, train);

	// asserts are needed because boost deallocates the pointer memory after being called...
	int expected_layer_tensors = 4;
	for (int i = 0; i < expected_layer_tensors; ++i) {
		//std::cout << "Layer batch size (" << i << "): " << model_interpreter.getLayerTensor(i)->getBatchSize() << std::endl;
		//std::cout << "Layer memory size (" << i << "): " << model_interpreter.getLayerTensor(i)->getMemorySize() << std::endl;
		//std::cout << "Layer memory size (" << i << "): " << model_interpreter.getLayerTensor(i)->getLayerSize() << std::endl;
		assert(model_interpreter.getLayerTensor(i)->getBatchSize() == batch_size); // sinks
		assert(model_interpreter.getLayerTensor(i)->getMemorySize() == memory_size + 1); // sinks
		if (i == 0) {
			assert(model_interpreter.getLayerTensor(i)->getLayerSize() == 2); // sinks
		}
		else if (i == 1) {
			assert(model_interpreter.getLayerTensor(i)->getLayerSize() == 3); // sources
		}
		else if (i == 2) {
			assert(model_interpreter.getLayerTensor(i)->getLayerSize() == 2); // sink
		}
		else if (i == 3) {
			assert(model_interpreter.getLayerTensor(i)->getLayerSize() == 1); // sources
		}
	}
	int expected_weight_tensors = 3;
	for (int i = 0; i < expected_weight_tensors; ++i) {
		//std::cout << "Weight Layer1 size (" << i << "): " << model_interpreter.getWeightTensor(i)->getLayer1Size() << std::endl;
		//std::cout << "Weight Layer1 size (" << i << "): " << model_interpreter.getWeightTensor(i)->getLayer2Size() << std::endl;
		//std::cout << "Weight NParams size (" << i << "): " << model_interpreter.getWeightTensor(i)->getNSolverParams() << std::endl;
		assert(model_interpreter.getWeightTensor(i)->getNSolverParams() == 3);
		if (i == 0) {
			assert(model_interpreter.getWeightTensor(i)->getLayer1Size() == 3);
			assert(model_interpreter.getWeightTensor(i)->getLayer2Size() == 2);
		}
		else if (i == 1) {
			assert(model_interpreter.getWeightTensor(i)->getLayer1Size() == 1);
			assert(model_interpreter.getWeightTensor(i)->getLayer2Size() == 2);
		}
		else if (i == 2) {
			assert(model_interpreter.getWeightTensor(i)->getLayer1Size() == 2);
			assert(model_interpreter.getWeightTensor(i)->getLayer2Size() == 2);
		}
	}
	std::vector<int> expected_operation_steps = { 1, 2 };
	for (int i = 0; i < expected_operation_steps.size(); ++i) {
		for (int j = 0; j < expected_operation_steps[i]; ++j) {
			//std::cout << "Source Layer Time Step (" << i << "): " << model_interpreter.getOperationSteps(i)[j].source_layer.time_step << std::endl;
			//std::cout << "Sink Layer Time Step (" << i << "): " << model_interpreter.getOperationSteps(i)[j].sink_layer.time_step << std::endl;
			assert(model_interpreter.getOperationSteps(i)[j].source_layer.time_step == 0);
			assert(model_interpreter.getOperationSteps(i)[j].sink_layer.time_step == 0);
			assert(model_interpreter.getOperationSteps(i)[j].sink_layer.integration->getName() == "SumTensorOp");
			assert(model_interpreter.getOperationSteps(i)[j].sink_layer.integration_error->getName() == "SumErrorTensorOp");
			assert(model_interpreter.getOperationSteps(i)[j].sink_layer.integration_weight_grad->getName() == "SumWeightGradTensorOp");
			assert(model_interpreter.getOperationSteps(i)[j].sink_layer.activation->getName() == "ReLUTensorOp");
			assert(model_interpreter.getOperationSteps(i)[j].sink_layer.activation_grad->getName() == "ReLUGradTensorOp");
			assert(model_interpreter.getOperationSteps(i)[j].weight.solver->getName() == "SGDTensorOp");
			if (j == 0) {
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.integration->getName() == "SumTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.integration_error->getName() == "SumErrorTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.integration_weight_grad->getName() == "SumWeightGradTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.activation->getName() == "LinearTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.activation_grad->getName() == "LinearGradTensorOp");
			}
			else if (i == 1 && j == 1) {
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.integration->getName() == "SumTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.integration_error->getName() == "SumErrorTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.integration_weight_grad->getName() == "SumWeightGradTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.activation->getName() == "ReLUTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.activation_grad->getName() == "ReLUGradTensorOp");
			}
			else {
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.integration->getName() == "SumTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.integration_error->getName() == "SumErrorTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.integration_weight_grad->getName() == "SumWeightGradTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.activation->getName() == "LinearTensorOp");
				assert(model_interpreter.getOperationSteps(i)[j].source_layer.activation_grad->getName() == "LinearGradTensorOp");
			}
		}
	}
}

void test_allocateModelErrorTensor()
{
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 2;

	model_interpreter.allocateModelErrorTensor(batch_size, memory_size);

	assert(model_interpreter.getModelError()->getBatchSize() == 4);
	assert(model_interpreter.getModelError()->getMemorySize() == 2);
}

void test_reInitNodes()
{
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 2;

	// TODO
}

void test_reInitModelError()
{
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 2;

	model_interpreter.allocateModelErrorTensor(batch_size, memory_size);
	Eigen::Tensor<float, 2> ones(batch_size, memory_size); ones.setConstant(1);
	model_interpreter.getModelError()->getError() = ones;
	assert(model_interpreter.getModelError()->getError()(0, 0) == 1);

	model_interpreter.reInitModelError();
	assert(model_interpreter.getModelError()->getError()(0, 0) == 0);
}

void test_mapValuesToLayers()
{
Model<float> model_mapValuesToLayers = makeModelToy1();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 1;
	const bool train = true;

	// initialize nodes
	// NOTE: input and biases have been activated when the model was created

	model_interpreter.getForwardPropogationOperations(model_mapValuesToLayers, batch_size, memory_size, train);

	// create the input
	const std::vector<std::string> node_ids = { "0", "1" };
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)node_ids.size());
	input.setValues({
		{{1, 5}},
		{{2, 6}},
		{{3, 7}},
		{{4, 8}} });

	auto node0 = model_mapValuesToLayers.getNode("0");
	auto node1 = model_mapValuesToLayers.getNode("1");

	model_interpreter.mapValuesToLayers(model_mapValuesToLayers, input, node_ids, "output");
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			assert(model_interpreter.getLayerTensor(node0.getTensorIndex().first)->getOutput()(i, j, node0.getTensorIndex().second) == input(i, j, 0));
			assert(model_interpreter.getLayerTensor(node1.getTensorIndex().first)->getOutput()(i, j, node1.getTensorIndex().second) == input(i, j, 1));
		}
	}

	model_interpreter.mapValuesToLayers(model_mapValuesToLayers, input, node_ids, "derivative");
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			assert(model_interpreter.getLayerTensor(node0.getTensorIndex().first)->getDerivative()(i, j, node0.getTensorIndex().second) == input(i, j, 0));
			assert(model_interpreter.getLayerTensor(node1.getTensorIndex().first)->getDerivative()(i, j, node1.getTensorIndex().second) == input(i, j, 1));
		}
	}

	model_interpreter.mapValuesToLayers(model_mapValuesToLayers, input, node_ids, "error");
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			assert(model_interpreter.getLayerTensor(node0.getTensorIndex().first)->getError()(i, j, node0.getTensorIndex().second) == input(i, j, 0));
			assert(model_interpreter.getLayerTensor(node1.getTensorIndex().first)->getError()(i, j, node1.getTensorIndex().second) == input(i, j, 1));
		}
	}

	model_interpreter.mapValuesToLayers(model_mapValuesToLayers, input, node_ids, "dt");
	for (int i = 0; i < batch_size; ++i) {
		for (int j = 0; j < memory_size; ++j) {
			assert(model_interpreter.getLayerTensor(node0.getTensorIndex().first)->getDt()(i, j, node0.getTensorIndex().second) == input(i, j, 0));
			assert(model_interpreter.getLayerTensor(node1.getTensorIndex().first)->getDt()(i, j, node1.getTensorIndex().second) == input(i, j, 1));
		}
	}
}

void test_executeForwardPropogationOperations()
{
Model<float> model_executeForwardPropogationOperations = makeModelToy1();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 1;
	const bool train = true;

	// compile the graph into a set of operations
	model_interpreter.getForwardPropogationOperations(model_executeForwardPropogationOperations, batch_size, memory_size, train);

	// create the input
	const std::vector<std::string> node_ids = { "0", "1" };
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)node_ids.size());
	input.setValues({
		{{1, 5}},
		{{2, 6}},
		{{3, 7}},
		{{4, 8}} });
	model_interpreter.mapValuesToLayers(model_executeForwardPropogationOperations, input, node_ids, "output");

	// create the bias
	model_interpreter.initBiases(model_executeForwardPropogationOperations);

	model_interpreter.executeForwardPropogationOperations(0);

	// test values of output nodes
	Eigen::Tensor<float, 2> output(batch_size, 2);
	output.setValues({ {15, 15}, {19, 19}, {23, 23}, {27, 27} });
	Eigen::Tensor<float, 2> net_input(batch_size, 2);
	net_input.setValues({ { 15, 15 },{ 19, 19 },{ 23, 23 },{ 27, 27 } });

	// Test
	const std::vector<std::string> output_nodes = { "4", "5" };
	auto nodes_map = model_executeForwardPropogationOperations.getNodesMap();

	cudaStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);
	for (int i = 0; i < (int)output_nodes.size(); ++i) {
		const std::string node_name = output_nodes[i];
		model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->syncHAndDInput(device);
		model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->syncHAndDOutput(device);
	}
	assert(cudaStreamSynchronize(stream) == cudaSuccess);
	assert(cudaStreamDestroy(stream) == cudaSuccess);

	for (int i = 0; i < (int)output_nodes.size(); ++i) {
		const std::string node_name = output_nodes[i];
		for (int j = 0; j < batch_size; ++j) {
			for (int k = 0; k < memory_size; ++k) {
				//std::cout << "Node: " << node_name << "; Batch: " << j << "; Memory: " << k << std::endl;
				//std::cout << "Calc Output: " << model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getOutput()(j, k, nodes_map.at(node_name)->getTensorIndex().second) << ", Expected Output: " << output(j, i) << std::endl;
				//std::cout << "Calc Net Input: " << model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getInput()(j, k, nodes_map.at(node_name)->getTensorIndex().second) << ", Expected Net Input: " << net_input(j, i) << std::endl;
				assert(model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getInput()(j, k, nodes_map.at(node_name)->getTensorIndex().second) == net_input(j, i));
				assert(model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getOutput()(j, k, nodes_map.at(node_name)->getTensorIndex().second) == output(j, i));
			}
		}
	}
}

void test_executeModelErrorOperations()
{
	Model<float> model_executeModelErrorOperations = makeModelToy1();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 1;
	const bool train = true;

	// compile the graph into a set of operations
	model_interpreter.getForwardPropogationOperations(model_executeModelErrorOperations, batch_size, memory_size, train);

	// create the input
	const std::vector<std::string> node_ids = { "0", "1" };
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)node_ids.size());
	input.setValues({
		{{1, 5}},
		{{2, 6}},
		{{3, 7}},
		{{4, 8}} });
	model_interpreter.mapValuesToLayers(model_executeModelErrorOperations, input, node_ids, "output");

	model_interpreter.initBiases(model_executeModelErrorOperations); // create the bias	
	model_interpreter.executeForwardPropogationOperations(0); // FP
	model_interpreter.allocateModelErrorTensor(batch_size, memory_size); // allocate the memory

	// calculate the model error
	std::vector<std::string> output_nodes = { "4", "5" };
	Eigen::Tensor<float, 2> expected(batch_size, (int)output_nodes.size());
	expected.setValues({ {0, 1}, {0, 1}, {0, 1}, {0, 1} });
	LossFunctionTensorOp<float, Eigen::GpuDevice>* solver = new MSETensorOp<float, Eigen::GpuDevice>();
	LossFunctionGradTensorOp<float, Eigen::GpuDevice>* solver_grad = new MSEGradTensorOp<float, Eigen::GpuDevice>();
	const int layer_id = model_executeModelErrorOperations.getNode("4").getTensorIndex().first;
	model_interpreter.executeModelErrorOperations(expected, layer_id, solver, solver_grad, 0);

	cudaStream_t stream; // The stream will be destroyed by GpuStreamDevice once the function goes out of scope!
	assert(cudaStreamCreateWithFlags(&stream, cudaStreamNonBlocking) == cudaSuccess);
	Eigen::GpuStreamDevice stream_device(&stream, 0);
	Eigen::GpuDevice device(&stream_device);
	auto nodes_map = model_executeModelErrorOperations.getNodesMap();
	for (int i = 0; i < (int)output_nodes.size(); ++i) {
		const std::string node_name = output_nodes[i];
		model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->syncHAndDError(device);
	}
	model_interpreter.getModelError()->syncHAndDError(device);
	assert(cudaStreamSynchronize(stream) == cudaSuccess);
	assert(cudaStreamDestroy(stream) == cudaSuccess);

	Eigen::Tensor<float, 2> error(batch_size, memory_size);
	error.setValues({ {105.25}, {171.25}, {253.25}, {351.25} });
	for (int j = 0; j < batch_size; ++j) {
		for (int k = 0; k < memory_size; ++k) {
			assert(model_interpreter.getModelError()->getError()(j, k) == error(j, k));
		}
	}

	// TODO: include full memory size
	Eigen::Tensor<float, 2> node_error(batch_size, (int)output_nodes.size());
	node_error.setValues({ {-7.5, -7}, {-9.5, -9}, {-11.5, -11}, {-13.5, -13} });
	for (int i = 0; i < (int)output_nodes.size(); ++i) {
		const std::string node_name = output_nodes[i];
		for (int j = 0; j < batch_size; ++j) {
			for (int k = 0; k < memory_size; ++k) {
				assert(model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getError()(j, k, nodes_map.at(node_name)->getTensorIndex().second) == node_error(j, i));
			}
		}
	}
}

void test_executeBackwardPropogationOperations()
{
Model<float> model_executeBackwardPropogationOperations = makeModelToy1();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 1;
	const bool train = true;

	// compile the graph into a set of operations
	model_interpreter.getForwardPropogationOperations(model_executeBackwardPropogationOperations, batch_size, memory_size, train);

	// create the input
	const std::vector<std::string> node_ids = { "0", "1" };
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)node_ids.size());
	input.setValues({
		{{1, 5}},
		{{2, 6}},
		{{3, 7}},
		{{4, 8}} });
	model_interpreter.mapValuesToLayers(model_executeBackwardPropogationOperations, input, node_ids, "output");

	model_interpreter.initBiases(model_executeBackwardPropogationOperations); // create the bias	
	model_interpreter.executeForwardPropogationOperations(0); // FP
	model_interpreter.allocateModelErrorTensor(batch_size, memory_size); // allocate the memory

	// calculate the model error
	std::vector<std::string> output_nodes = { "4", "5" };
	Eigen::Tensor<float, 2> expected(batch_size, (int)output_nodes.size());
	expected.setValues({ {0, 1}, {0, 1}, {0, 1}, {0, 1} });
	LossFunctionTensorOp<float, Eigen::GpuDevice>* solver = new MSETensorOp<float, Eigen::GpuDevice>();
	LossFunctionGradTensorOp<float, Eigen::GpuDevice>* solver_grad = new MSEGradTensorOp<float, Eigen::GpuDevice>();
	const int layer_id = model_executeBackwardPropogationOperations.getNode("4").getTensorIndex().first;
	model_interpreter.executeModelErrorOperations(expected, layer_id, solver, solver_grad, 0);

	model_interpreter.executeBackwardPropogationOperations(0); // BP

	std::vector<std::string> error_nodes = { "6", "2", "3" };
	Eigen::Tensor<float, 2> error(batch_size, (int)error_nodes.size());
	error.setValues({ {-29, -14.5, -14.5}, {-37, -18.5, -18.5}, {-45, -22.5, -22.5}, {-53, -26.5, -26.5} });
	Eigen::Tensor<float, 2> derivative(batch_size, (int)error_nodes.size());
	derivative.setValues({ {1, 1, 1}, {1, 1, 1}, {1, 1, 1}, {1, 1, 1} });
	auto nodes_map = model_executeBackwardPropogationOperations.getNodesMap();
	for (int i = 0; i < (int)error_nodes.size(); ++i) {
		const std::string node_name = error_nodes[i];
		for (int j = 0; j < batch_size; ++j) {
			for (int k = 0; k < memory_size; ++k) {
				assert(model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getError()(j, k, nodes_map.at(node_name)->getTensorIndex().second) == error(j, i));
				assert(model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getDerivative()(j, k, nodes_map.at(node_name)->getTensorIndex().second) == derivative(j, i));
			}
		}
	}
}

void test_executeWeightErrorOperations()
{
	Model<float> model_executeWeightErrorOperations = makeModelToy1();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 1;
	const bool train = true;

	// compile the graph into a set of operations
	model_interpreter.getForwardPropogationOperations(model_executeWeightErrorOperations, batch_size, memory_size, train);

	// create the input
	const std::vector<std::string> node_ids = { "0", "1" };
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)node_ids.size());
	input.setValues({
		{{1, 5}},
		{{2, 6}},
		{{3, 7}},
		{{4, 8}} });
	model_interpreter.mapValuesToLayers(model_executeWeightErrorOperations, input, node_ids, "output");

	model_interpreter.initBiases(model_executeWeightErrorOperations); // create the bias	
	model_interpreter.executeForwardPropogationOperations(0); // FP
	model_interpreter.allocateModelErrorTensor(batch_size, memory_size); // allocate the memory

	// calculate the model error
	std::vector<std::string> output_nodes = { "4", "5" };
	Eigen::Tensor<float, 2> expected(batch_size, (int)output_nodes.size());
	expected.setValues({ {0, 1}, {0, 1}, {0, 1}, {0, 1} });
	LossFunctionTensorOp<float, Eigen::GpuDevice>* solver = new MSETensorOp<float, Eigen::GpuDevice>();
	LossFunctionGradTensorOp<float, Eigen::GpuDevice>* solver_grad = new MSEGradTensorOp<float, Eigen::GpuDevice>();
	const int layer_id = model_executeWeightErrorOperations.getNode("4").getTensorIndex().first;
	model_interpreter.executeModelErrorOperations(expected, layer_id, solver, solver_grad, 0);

	model_interpreter.executeBackwardPropogationOperations(0); // BP
	model_interpreter.executeWeightErrorOperations(); // Weight error

	// test values of input and hidden layers
	const std::vector<std::string> weight_ids = { "0", "1", "2", "3", "4", "5", "6", "7", "8", "9", "10", "11" };
	Eigen::Tensor<float, 1> weights((int)weight_ids.size());
	weights.setValues({ 56.25f, 56.25f, 138.25f, 138.25f, 20.5f, 20.5f,
		110.0f, 105.0f, 110.0f, 105.0f, 10.5f, 10.0f });
	auto weights_map = model_executeWeightErrorOperations.getWeightsMap();
	for (int i = 0; i < weight_ids.size(); ++i)
	{
		//std::cout << "Weight Error: " << weight_ids[i] << "; Calculated: " << model_interpreter.getWeightTensor(
		//	std::get<0>(weights_map.at(weight_ids[i])->getTensorIndex()[0]))->getError()(
		//		std::get<1>(weights_map.at(weight_ids[i])->getTensorIndex()[0]), std::get<2>(weights_map.at(weight_ids[i])->getTensorIndex()[0])) << ", Expected: " << weights(i) << std::endl;
		assert(model_interpreter.getWeightTensor(
			std::get<0>(weights_map.at(weight_ids[i])->getTensorIndex()[0]))->getError()(
				std::get<1>(weights_map.at(weight_ids[i])->getTensorIndex()[0]), std::get<2>(weights_map.at(weight_ids[i])->getTensorIndex()[0])) == weights(i));
	}
}

void test_executeWeightUpdateOperations()
{
	Model<float> model_executeWeightUpdateOperations = makeModelToy1();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 1;
	const bool train = true;

	// compile the graph into a set of operations
	model_interpreter.getForwardPropogationOperations(model_executeWeightUpdateOperations, batch_size, memory_size, train);

	// create the input
	const std::vector<std::string> node_ids = { "0", "1" };
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)node_ids.size());
	input.setValues({
		{{1, 5}},
		{{2, 6}},
		{{3, 7}},
		{{4, 8}} });
	model_interpreter.mapValuesToLayers(model_executeWeightUpdateOperations, input, node_ids, "output");

	model_interpreter.initBiases(model_executeWeightUpdateOperations); // create the bias	
	model_interpreter.executeForwardPropogationOperations(0); // FP
	model_interpreter.allocateModelErrorTensor(batch_size, memory_size); // allocate the memory

	// calculate the model error
	std::vector<std::string> output_nodes = { "4", "5" };
	Eigen::Tensor<float, 2> expected(batch_size, (int)output_nodes.size());
	expected.setValues({ {0, 1}, {0, 1}, {0, 1}, {0, 1} });
	LossFunctionTensorOp<float, Eigen::GpuDevice>* solver = new MSETensorOp<float, Eigen::GpuDevice>();
	LossFunctionGradTensorOp<float, Eigen::GpuDevice>* solver_grad = new MSEGradTensorOp<float, Eigen::GpuDevice>();
	const int layer_id = model_executeWeightUpdateOperations.getNode("4").getTensorIndex().first;
	model_interpreter.executeModelErrorOperations(expected, layer_id, solver, solver_grad, 0);

	model_interpreter.executeBackwardPropogationOperations(0); // BP
	model_interpreter.executeWeightErrorOperations(); // Weight error
	model_interpreter.executeWeightUpdateOperations(); // Weight update

	// test values of input and hidden layers
	const std::vector<std::string> weight_ids = { "0", "1", "2", "3", "4", "5", "6", "7", "8", "9", "10", "11" };
	Eigen::Tensor<float, 1> weights((int)weight_ids.size());
	weights.setValues({ 0.4375f, 0.4375f, -0.382499933f, -0.382499933f, 0.795000017f, 0.795000017f,
		-0.100000024f, -0.0499999523f, -0.100000024, -0.0499999523f, 0.894999981f, 0.899999976f });
	auto weights_map = model_executeWeightUpdateOperations.getWeightsMap();
	for (int i = 0; i < weight_ids.size(); ++i)
	{
		//std::cout<<"Weight: "<< weight_ids[i] <<"; Calculated: "<<model_interpreter.getWeightTensor(
		//	std::get<0>(weights_map.at(weight_ids[i])->getTensorIndex()[0]))->getWeight()(
		//	std::get<1>(weights_map.at(weight_ids[i])->getTensorIndex()[0]), std::get<2>(weights_map.at(weight_ids[i])->getTensorIndex()[0])) <<", Expected: "<<weights(i)<<std::endl;
		assert(model_interpreter.getWeightTensor(
			std::get<0>(weights_map.at(weight_ids[i])->getTensorIndex()[0]))->getWeight()(
				std::get<1>(weights_map.at(weight_ids[i])->getTensorIndex()[0]), std::get<2>(weights_map.at(weight_ids[i])->getTensorIndex()[0])) == weights(i));
	}
}

void test_modelTrainer1()
{
	Model<float> model_modelTrainer1 = makeModelToy1();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 4;
	const int memory_size = 1;
	const bool train = true;

	// update the model solver
	std::shared_ptr<SolverOp<float>> solver(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
	for (auto& weight_map : model_modelTrainer1.getWeightsMap()) {
		if (weight_map.second->getSolverOp()->getName() == "SGDOp")
			weight_map.second->setSolverOp(solver);
	}

	// compile the graph into a set of operations and allocate all tensors
	model_interpreter.getForwardPropogationOperations(model_modelTrainer1, batch_size, memory_size, train);
	model_interpreter.allocateModelErrorTensor(batch_size, memory_size);

	// create the input
	const std::vector<std::string> node_ids = { "0", "1" };
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)node_ids.size());
	input.setValues({
		{{1, 5}},
		{{2, 6}},
		{{3, 7}},
		{{4, 8}} });

	// create the expected output
	std::vector<std::string> output_nodes = { "4", "5" };
	Eigen::Tensor<float, 2> expected(batch_size, (int)output_nodes.size());
	expected.setValues({ {0, 1}, {0, 1}, {0, 1}, {0, 1} });
	LossFunctionTensorOp<float, Eigen::GpuDevice>* loss_function = new MSETensorOp<float, Eigen::GpuDevice>();
	LossFunctionGradTensorOp<float, Eigen::GpuDevice>* loss_function_grad = new MSEGradTensorOp<float, Eigen::GpuDevice>();
	const int layer_id = model_modelTrainer1.getNode("4").getTensorIndex().first;

	// iterate until we find the optimal values
	const int max_iter = 20;
	for (int iter = 0; iter < max_iter; ++iter)
	{
		// assign the input data
		model_interpreter.mapValuesToLayers(model_modelTrainer1, input, node_ids, "output");
		model_interpreter.initBiases(model_modelTrainer1); // create the bias	

		model_interpreter.executeForwardPropogationOperations(0); //FP

		// calculate the model error and node output error
		model_interpreter.executeModelErrorOperations(expected, layer_id, loss_function, loss_function_grad, 0);
		std::cout << "Error at iteration: " << iter << " is " << model_interpreter.getModelError()->getError().sum() << std::endl;

		model_interpreter.executeBackwardPropogationOperations(0); // BP
		model_interpreter.executeWeightErrorOperations(); // Weight error
		model_interpreter.executeWeightUpdateOperations(); // Weight update

		// reinitialize the model
		if (iter != max_iter - 1) {
			model_interpreter.reInitNodes();
			model_interpreter.reInitModelError();
		}
	}

	const Eigen::Tensor<float, 0> total_error = model_interpreter.getModelError()->getError().sum();
	assert(total_error(0) <= 757.0);
}

Model<float> makeModelToy2()
{
	/**
	 * Directed Cyclic Graph Toy Network Model
	*/
	Node<float> i1, h1, o1, b1, b2;
	Link l1, l2, l3, lb1, lb2;
	Weight<float> w1, w2, w3, wb1, wb2;
	Model<float> model2;
	// Toy network: 1 hidden layer, fully connected, DCG
	i1 = Node<float>("0", NodeType::input, NodeStatus::activated, std::shared_ptr<ActivationOp<float>>(new LinearOp<float>()), std::shared_ptr<ActivationOp<float>>(new LinearGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	h1 = Node<float>("1", NodeType::hidden, NodeStatus::initialized, std::shared_ptr<ActivationOp<float>>(new ReLUOp<float>()), std::shared_ptr<ActivationOp<float>>(new ReLUGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	o1 = Node<float>("2", NodeType::output, NodeStatus::initialized, std::shared_ptr<ActivationOp<float>>(new ReLUOp<float>()), std::shared_ptr<ActivationOp<float>>(new ReLUGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	b1 = Node<float>("3", NodeType::bias, NodeStatus::activated, std::shared_ptr<ActivationOp<float>>(new LinearOp<float>()), std::shared_ptr<ActivationOp<float>>(new LinearGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	b2 = Node<float>("4", NodeType::bias, NodeStatus::activated, std::shared_ptr<ActivationOp<float>>(new LinearOp<float>()), std::shared_ptr<ActivationOp<float>>(new LinearGradOp<float>()), std::shared_ptr<IntegrationOp<float>>(new SumOp<float>()), std::shared_ptr<IntegrationErrorOp<float>>(new SumErrorOp<float>()), std::shared_ptr<IntegrationWeightGradOp<float>>(new SumWeightGradOp<float>()));
	// weights  
	std::shared_ptr<WeightInitOp<float>> weight_init;
	std::shared_ptr<SolverOp<float>> solver;
	// weight_init.reset(new RandWeightInitOp(1.0)); // No random init for testing
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	w1 = Weight<float>("0", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	w2 = Weight<float>("1", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	w3 = Weight<float>("2", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	wb1 = Weight<float>("3", weight_init, solver);
	weight_init.reset(new ConstWeightInitOp<float>(1.0));
	solver.reset(new SGDOp<float>(0.01, 0.9));
	wb2 = Weight<float>("4", weight_init, solver);
	weight_init.reset();
	solver.reset();
	// links
	l1 = Link("0", "0", "1", "0");
	l2 = Link("1", "1", "2", "1");
	l3 = Link("2", "1", "1", "2"); // cycle
	lb1 = Link("3", "3", "1", "3");
	lb2 = Link("4", "4", "2", "4");
	model2.setId(2);
	model2.addNodes({ i1, h1, o1, b1, b2 });
	model2.addWeights({ w1, w2, w3, wb1, wb2 });
	model2.addLinks({ l1, l2, l3, lb1, lb2 });
	model2.findCycles();
	return model2;
}

void test_FPTT()
{
	Model<float> model_FPTT = makeModelToy2();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 5;
	const int memory_size = 8;
	const bool train = true;

	// compile the graph into a set of operations and allocate all tensors
	model_interpreter.getForwardPropogationOperations(model_FPTT, batch_size, memory_size, train);
	model_interpreter.allocateModelErrorTensor(batch_size, memory_size);

	// create the input
	const std::vector<std::string> input_ids = { "0", "3", "4" }; // biases are set to zero
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)input_ids.size());
	input.setValues(
		{ {{8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}, {1, 0, 0}},
		{{9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}},
		{{10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}},
		{{11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}},
		{{12, 0, 0}, {11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}} }
	);
	model_interpreter.mapValuesToLayers(model_FPTT, input, input_ids, "output");

	model_interpreter.FPTT(4);

	// test values of output nodes
	Eigen::Tensor<float, 3> output(batch_size, memory_size, 5); // dim2: # of model nodes
	output.setValues({
		{{8, 26, 26, 0, 0}, {7, 18, 18, 0, 0}, {6, 11, 11, 0, 0}, {5, 5, 5, 0, 0}, {4, 0, 0, 0, 0}, {3, 0, 0, 0, 0}, {2, 0, 0, 0, 0}, {1, 0, 0, 0, 0}},
		{{9, 30, 30, 0, 0}, {8, 21, 21, 0, 0}, {7, 13, 13, 0, 0}, {6, 6, 6, 0, 0}, {5, 0, 0, 0, 0}, {4, 0, 0, 0, 0}, {3, 0, 0, 0, 0}, {2, 0, 0, 0, 0}},
		{{10, 34, 34, 0, 0}, {9, 24, 24, 0, 0}, {8, 15, 15, 0, 0}, {7, 7, 7, 0, 0}, {6, 0, 0, 0, 0}, {5, 0, 0, 0, 0}, {4, 0, 0, 0, 0}, {3, 0, 0, 0, 0}},
		{{11, 38, 38, 0, 0}, {10, 27, 27, 0, 0}, {9, 17, 17, 0, 0}, {8, 8, 8, 0, 0}, {7, 0, 0, 0, 0}, {6, 0, 0, 0, 0}, {5, 0, 0, 0, 0}, {4, 0, 0, 0, 0}},
		{{12, 42, 42, 0, 0}, {11, 30, 30, 0, 0}, {10, 19, 19, 0, 0}, {9, 9, 9, 0, 0}, {8, 0, 0, 0, 0}, {7, 0, 0, 0, 0}, {6, 0, 0, 0, 0}, {5, 0, 0, 0, 0}} }
	);
	Eigen::Tensor<float, 3> net_input(batch_size, memory_size, 5); // dim2: # of model nodes
	net_input.setValues({
		{{0, 26, 26, 0, 0}, {0, 18, 18, 0, 0}, {0, 11, 11, 0, 0}, {0, 5, 5, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}},
		{{0, 30, 30, 0, 0}, {0, 21, 21, 0, 0}, {0, 13, 13, 0, 0}, {0, 6, 6, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}},
		{{0, 34, 34, 0, 0}, {0, 24, 24, 0, 0}, {0, 15, 15, 0, 0}, {0, 7, 7, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}},
		{{0, 38, 38, 0, 0}, {0, 27, 27, 0, 0}, {0, 17, 17, 0, 0}, {0, 8, 8, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}},
		{{0, 42, 42, 0, 0}, {0, 30, 30, 0, 0}, {0, 19, 19, 0, 0}, {0, 9, 9, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}} }
	);
	const std::vector<std::string> output_nodes = { "0", "1", "2", "3", "4" };

	auto nodes_map = model_FPTT.getNodesMap();
	for (int j = 0; j < batch_size; ++j) {
		for (int k = 0; k < memory_size; ++k) {
			for (int i = 0; i < output_nodes.size(); ++i) {
				const std::string node_name = output_nodes[i];
				//std::cout << "Node: " << node_name << "; Batch: " << j << "; Memory: " << k << std::endl;
				//std::cout << "Calc Output: " << model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getOutput()(j, k, nodes_map.at(node_name)->getTensorIndex().second) << ", Expected Output: " << output(j, k, i) << std::endl;
				//std::cout << "Calc Net Input: " << model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getInput()(j, k, nodes_map.at(node_name)->getTensorIndex().second) << ", Expected Net Input: " << net_input(j, k, i) << std::endl;
				assert(model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getOutput()(j, k, nodes_map.at(node_name)->getTensorIndex().second) == output(j, k, i));
				assert(model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getInput()(j, k, nodes_map.at(node_name)->getTensorIndex().second) == net_input(j, k, i));
			}
		}
	}
}

void test_CETT()
{
	Model<float> model_CETT = makeModelToy2();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 5;
	const int memory_size = 8;
	const bool train = true;

	// compile the graph into a set of operations and allocate all tensors
	model_interpreter.getForwardPropogationOperations(model_CETT, batch_size, memory_size, train);
	model_interpreter.allocateModelErrorTensor(batch_size, memory_size);

	// create the input
	const std::vector<std::string> input_ids = { "0", "3", "4" };  // biases are set to zero
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)input_ids.size());
	input.setValues(
		{ {{8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}, {1, 0, 0}},
		{{9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}},
		{{10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}},
		{{11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}},
		{{12, 0, 0}, {11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}} }
	);
	model_interpreter.mapValuesToLayers(model_CETT, input, input_ids, "output");

	model_interpreter.FPTT(4);

	// calculate the error
	// expected output (from t=n to t=0)
	const std::vector<std::string> output_nodes = { "2" };
	// y = m1*(m2*x + b*yprev) where m1 = 1, m2 = 1 and b = -1
	Eigen::Tensor<float, 3> expected(batch_size, memory_size, (int)output_nodes.size());
	expected.setValues(
		{ { { 1 },{ 1 },{ 2 },{ 2 },{ 3 },{ 3 },{ 4 },{ 4 } },
		{ { 1 },{ 2 },{ 2 },{ 3 },{ 3 },{ 4 },{ 4 },{ 5 } },
		{ { 2 },{ 2 },{ 3 },{ 3 },{ 4 },{ 4 },{ 5 },{ 5 } },
		{ { 2 },{ 3 },{ 3 },{ 4 },{ 4 },{ 5 },{ 5 },{ 6 } },
		{ { 3 },{ 3 },{ 4 },{ 4 },{ 5 },{ 5 },{ 6 },{ 6 } } }
	);
	LossFunctionOp<float>* loss_function = new MSEOp<float>();
	LossFunctionGradOp<float>* loss_function_grad = new MSEGradOp<float>();
	model_interpreter.CETT(model_CETT, expected, output_nodes, loss_function, loss_function_grad, 4);

	// test values of errors of the output nodes
	Eigen::Tensor<float, 2> model_error(batch_size, memory_size);
	model_error.setValues({
		{242,98,32,2,0,0,0,0},
		{312.5f,144.5f,40.5f,4.5f,0,0,0,0},
		{420.5f,180.5f,60.5f,4.5f,0,0,0,0},
		{512,242,72,8,0,0,0,0},
		{648,288,98,8,0,0,0,0} });
	Eigen::Tensor<float, 3> node_error(batch_size, memory_size, (int)output_nodes.size());
	node_error.setValues(
		{ { { -22 }, { -14 }, { -8 }, { -2 }, { 0.0f }, { 0.0f }, { 0.0f }, { 0.0f }},
			{ { -25 },{ -17 },{ -9 },{ -3 },{ 0.0f },{ 0.0f },{ 0.0f },{ 0.0f } },
			{ { -29 },{ -19 },{ -11 },{ -3 },{ 0.0f },{ 0.0f },{ 0.0f },{ 0.0f } },
			{ { -32 },{ -22 },{ -12 },{ -4 },{ 0.0f },{ 0.0f },{ 0.0f },{ 0.0f } },
			{ { -36 },{ -24 },{ -14 },{ -4 },{ 0.0f },{ 0.0f },{ 0.0f },{ 0.0f } } }
	);

	auto nodes_map = model_CETT.getNodesMap();
	for (int j = 0; j < batch_size; ++j) {
		for (int k = 0; k < memory_size; ++k) {
			//std::cout << "Batch: " << j << "; Memory: " << k << std::endl;
			//std::cout << "Calc Model Error: " << model_interpreter.getModelError()->getError()(j, k) << ", Expected Error: " << model_error(j, k) << std::endl;
			assert(model_interpreter.getModelError()->getError()(j, k), model_error(j, k), 1e-6);
			for (int i = 0; i < output_nodes.size(); ++i) {
				const std::string node_name = output_nodes[i];
				//std::cout << "Node: " << node_name << "; Batch: " << j << "; Memory: " << k << std::endl;
				//std::cout << "Calc Node Error: " << model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getError()(j, k, nodes_map.at(node_name)->getTensorIndex().second) << ", Expected Error: " << node_error(j, k, i) << std::endl;
				assert(model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getError()(j, k, nodes_map.at(node_name)->getTensorIndex().second) == node_error(j, k, i));
			}
		}
	}
}

void test_TBPTT()
{
	Model<float> model_TBPTT = makeModelToy2();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 5;
	const int memory_size = 8;
	const bool train = true;

	// compile the graph into a set of operations and allocate all tensors
	model_interpreter.getForwardPropogationOperations(model_TBPTT, batch_size, memory_size, train);
	model_interpreter.allocateModelErrorTensor(batch_size, memory_size);

	// create the input
	const std::vector<std::string> input_ids = { "0", "3", "4" };  // biases are set to zero
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)input_ids.size());
	input.setValues(
		{ {{8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}, {1, 0, 0}},
		{{9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}},
		{{10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}},
		{{11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}},
		{{12, 0, 0}, {11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}} }
	);
	model_interpreter.mapValuesToLayers(model_TBPTT, input, input_ids, "output");

	model_interpreter.FPTT(4);

	// calculate the error
	// expected output (from t=n to t=0)
	const std::vector<std::string> output_nodes = { "2" };
	// y = m1*(m2*x + b*yprev) where m1 = 1, m2 = 1 and b = -1
	Eigen::Tensor<float, 3> expected(batch_size, memory_size, (int)output_nodes.size());
	expected.setValues(
		{ { { 1 },{ 1 },{ 2 },{ 2 },{ 3 },{ 3 },{ 4 },{ 4 } },
		{ { 1 },{ 2 },{ 2 },{ 3 },{ 3 },{ 4 },{ 4 },{ 5 } },
		{ { 2 },{ 2 },{ 3 },{ 3 },{ 4 },{ 4 },{ 5 },{ 5 } },
		{ { 2 },{ 3 },{ 3 },{ 4 },{ 4 },{ 5 },{ 5 },{ 6 } },
		{ { 3 },{ 3 },{ 4 },{ 4 },{ 5 },{ 5 },{ 6 },{ 6 } } }
	);
	LossFunctionOp<float>* loss_function = new MSEOp<float>();
	LossFunctionGradOp<float>* loss_function_grad = new MSEGradOp<float>();
	model_interpreter.CETT(model_TBPTT, expected, output_nodes, loss_function, loss_function_grad, 4);

	model_interpreter.TBPTT(4);

	// test values of output nodes
	Eigen::Tensor<float, 3> node_error(batch_size, memory_size, 5); // dim2: # of model nodes
	node_error.setValues({
		{ { -22, -22, -22, -22, -22 },{-36, -36, -14, -36, -14 },{ -44, -44, -8, -44, -8 },{ -46, -46, -2, -46, -2 },{ 0, -46, 0, 0, 0 },{ 0, 0, 0, 0, 0 },{ 0, 0, 0, 0, 0 },{ 0, 0, 0, 0, 0 } },
		{ { -25, -25, -25, -25, -25 },{ -42, -42, -17, -42, -17 },{ -51, -51, -9, -51, -9 },{ -54, -54, -3, -54, -3 },{ 0, -54, 0, 0, 0 },{ 0, 0, 0, 0, 0 },{ 0, 0, 0, 0, 0 },{ 0, 0, 0, 0, 0 } },
		{ { -29, -29, -29, -29, -29 },{ -48, -48, -19, -48, -19 },{ -59, -59, -11, -59, -11 },{ -62, -62, -3, -62, -3 },{ 0, -62, 0, 0, 0 },{ 0, 0, 0, 0, 0 },{ 0, 0, 0, 0, 0 },{ 0, 0, 0, 0, 0 } },
		{ { -32, -32, -32, -32, -32 },{ -54, -54, -22, -54, -22 },{ -66, -66, -12, -66, -12 },{ -70, -70, -4, -70, -4 },{ 0, -70, 0, 0, 0 },{ 0, 0, 0, 0, 0 },{ 0, 0, 0, 0, 0 },{ 0, 0, 0, 0, 0 } },
		{ {-36, -36, -36, -36, -36 },{-60, -60, -24, -60, -24 },{-74, -74, -14, -74, -14 },{ -78, -78, -4, -78, -4 },{ 0, -78, 0, 0, 0 },{ 0, 0, 0, 0, 0 },{ 0, 0, 0, 0, 0 },{ 0, 0, 0, 0, 0 } } }
	);
	Eigen::Tensor<float, 3> derivative(batch_size, memory_size, 5);
	derivative.setValues({
		{{1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}},
		{{1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}},
		{{1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}},
		{{1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}},
		{{1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {1, 1, 0, 1, 1}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}, {0, 0, 0, 0, 0}} }
	);
	const std::vector<std::string> error_nodes = { "0", "1", "2", "3", "4" };

	auto nodes_map = model_TBPTT.getNodesMap();
	for (int j = 0; j < batch_size; ++j) {
		for (int k = 0; k < memory_size; ++k) {
			for (int i = 0; i < error_nodes.size(); ++i) {
				const std::string node_name = error_nodes[i];
				//std::cout << "Node: " << node_name << "; Batch: " << j << "; Memory: " << k << std::endl;
				//std::cout << "Calc Error: " << model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getError()(j, k, nodes_map.at(node_name)->getTensorIndex().second) << ", Expected Error: " << node_error(j, k, i) << std::endl;
				//std::cout << "Calc Derivative: " << model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getDerivative()(j, k, nodes_map.at(node_name)->getTensorIndex().second) << ", Expected Derivative: " << derivative(j, k, i) << std::endl;
				assert(model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getError()(j, k, nodes_map.at(node_name)->getTensorIndex().second) == node_error(j, k, i));
				assert(model_interpreter.getLayerTensor(nodes_map.at(node_name)->getTensorIndex().first)->getDerivative()(j, k, nodes_map.at(node_name)->getTensorIndex().second) == derivative(j, k, i));
			}
		}
	}
}

void test_updateWeights()
{
	Model<float> model_updateWeights = makeModelToy2();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 5;
	const int memory_size = 8;
	const bool train = true;

	// compile the graph into a set of operations and allocate all tensors
	model_interpreter.getForwardPropogationOperations(model_updateWeights, batch_size, memory_size, train);
	model_interpreter.allocateModelErrorTensor(batch_size, memory_size);

	// create the input
	const std::vector<std::string> input_ids = { "0", "3", "4" };  // biases are set to zero
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)input_ids.size());
	input.setValues(
		{ {{8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}, {1, 0, 0}},
		{{9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}},
		{{10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}},
		{{11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}},
		{{12, 0, 0}, {11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}} }
	);
	model_interpreter.mapValuesToLayers(model_updateWeights, input, input_ids, "output");

	model_interpreter.FPTT(4);

	// calculate the error
	// expected output (from t=n to t=0)
	const std::vector<std::string> output_nodes = { "2" };
	// y = m1*(m2*x + b*yprev) where m1 = 1, m2 = 1 and b = -1
	Eigen::Tensor<float, 3> expected(batch_size, memory_size, (int)output_nodes.size());
	expected.setValues(
		{ { { 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 },{ 1 } },
		{ { 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 } },
		{ { 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 } },
		{ { 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 } },
		{ { 6 },{ 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 } } });
	LossFunctionOp<float>* loss_function = new MSEOp<float>();
	LossFunctionGradOp<float>* loss_function_grad = new MSEGradOp<float>();
	model_interpreter.CETT(model_updateWeights, expected, output_nodes, loss_function, loss_function_grad, 4);

	model_interpreter.TBPTT(4);
	model_interpreter.updateWeights();

	auto weights_map = model_updateWeights.getWeightsMap();
	// test values of output nodes
	std::vector<std::string> weight_ids = { "0", "1", "2", "3", "4" };
	Eigen::Tensor<float, 1> weights(weight_ids.size());
	weights.setValues({ -19.624f, -15.744f, -34.572f, 1.0f, 1.0f });
	for (int i = 0; i < weight_ids.size(); ++i) {
		assert(model_interpreter.getWeightTensor(
			std::get<0>(weights_map.at(weight_ids[i])->getTensorIndex()[0]))->getWeight()(
				std::get<1>(weights_map.at(weight_ids[i])->getTensorIndex()[0]), std::get<2>(weights_map.at(weight_ids[i])->getTensorIndex()[0])) == weights(i));
	}
}

void test_modelTrainer2()
{
	Model<float> model_modelTrainer2 = makeModelToy2();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 5;
	const int memory_size = 8;
	const bool train = true;

	// update the model solver
	std::shared_ptr<SolverOp<float>> solver(new AdamOp<float>(0.001, 0.9, 0.999, 1e-8));
	for (auto& weight_map : model_modelTrainer2.getWeightsMap()) {
		if (weight_map.second->getSolverOp()->getName() == "SGDOp")
			weight_map.second->setSolverOp(solver);
	}

	// compile the graph into a set of operations and allocate all tensors
	model_interpreter.getForwardPropogationOperations(model_modelTrainer2, batch_size, memory_size, train);
	model_interpreter.allocateModelErrorTensor(batch_size, memory_size);

	// create the input
	const std::vector<std::string> input_nodes = { "0", "3", "4" };  // biases are set to zero
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)input_nodes.size());
	input.setValues(
		{ {{8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}, {1, 0, 0}},
		{{9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}},
		{{10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}},
		{{11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}},
		{{12, 0, 0}, {11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}} }
	);

	// expected output (from t=n to t=0) for  y = m1*(m2*x + b*yprev) where m1 = 1, m2 = 1 and b = -1
	const std::vector<std::string> output_nodes = { "2" };
	Eigen::Tensor<float, 3> expected(batch_size, memory_size, (int)output_nodes.size());
	expected.setValues(
		{ { { 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 },{ 1 } },
		{ { 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 } },
		{ { 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 } },
		{ { 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 } },
		{ { 6 },{ 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 } } });
	LossFunctionOp<float>* loss_function = new MSEOp<float>();
	LossFunctionGradOp<float>* loss_function_grad = new MSEGradOp<float>();

	// iterate until we find the optimal values
	const int max_iter = 50;
	for (int iter = 0; iter < max_iter; ++iter)
	{
		// assign the input data
		model_interpreter.initBiases(model_modelTrainer2); // create the bias	
		model_interpreter.mapValuesToLayers(model_modelTrainer2, input, input_nodes, "output");

		model_interpreter.FPTT(4); //FP

		// calculate the model error and node output error
		model_interpreter.CETT(model_modelTrainer2, expected, output_nodes, loss_function, loss_function_grad, 4);
		std::cout << "Error at iteration: " << iter << " is " << model_interpreter.getModelError()->getError().sum() << std::endl;

		model_interpreter.TBPTT(4); // BP
		model_interpreter.updateWeights(); // Weight update

		// reinitialize the model
		if (iter != max_iter - 1) {
			model_interpreter.reInitNodes();
			model_interpreter.reInitModelError();
		}
	}

	const Eigen::Tensor<float, 0> total_error = model_interpreter.getModelError()->getError().sum();
	assert(total_error(0) <= 1492.6);
}

void test_getModelResults()
{
	Model<float> model_getModelResults = makeModelToy2();
	ModelInterpreterGpu<float> model_interpreter;
	const int batch_size = 5;
	const int memory_size = 8;
	const bool train = true;

	// compile the graph into a set of operations and allocate all tensors
	model_interpreter.getForwardPropogationOperations(model_getModelResults, batch_size, memory_size, train);
	model_interpreter.allocateModelErrorTensor(batch_size, memory_size);

	// create the input
	const std::vector<std::string> input_ids = { "0", "3", "4" };  // biases are set to zero
	Eigen::Tensor<float, 3> input(batch_size, memory_size, (int)input_ids.size());
	input.setValues(
		{ {{8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}, {1, 0, 0}},
		{{9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}, {2, 0, 0}},
		{{10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}, {3, 0, 0}},
		{{11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}, {4, 0, 0}},
		{{12, 0, 0}, {11, 0, 0}, {10, 0, 0}, {9, 0, 0}, {8, 0, 0}, {7, 0, 0}, {6, 0, 0}, {5, 0, 0}} }
	);
	model_interpreter.mapValuesToLayers(model_getModelResults, input, input_ids, "output");

	model_interpreter.FPTT(4);

	// calculate the error
	// expected output (from t=n to t=0)
	const std::vector<std::string> output_nodes = { "2" };
	// y = m1*(m2*x + b*yprev) where m1 = 1, m2 = 1 and b = -1
	Eigen::Tensor<float, 3> expected(batch_size, memory_size, (int)output_nodes.size());
	expected.setValues(
		{ { { 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 },{ 1 } },
		{ { 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 },{ 1 } },
		{ { 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 },{ 2 } },
		{ { 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 },{ 2 } },
		{ { 6 },{ 6 },{ 5 },{ 5 },{ 4 },{ 4 },{ 3 },{ 3 } } });
	LossFunctionOp<float>* loss_function = new MSEOp<float>();
	LossFunctionGradOp<float>* loss_function_grad = new MSEGradOp<float>();
	model_interpreter.CETT(model_getModelResults, expected, output_nodes, loss_function, loss_function_grad, 4);

	model_interpreter.TBPTT(4);
	model_interpreter.updateWeights();

	model_interpreter.getModelResults(model_getModelResults);

	// test values of output nodes
	Eigen::Tensor<float, 3> output(batch_size, memory_size, (int)output_nodes.size()); // dim2: # of model nodes
	output.setValues({
		{{26}, {18}, {11}, {5}, {0}, {0}, {0}, {0}},
		{{30}, {21}, {13}, {6}, {0}, {0}, {0}, {0}},
		{{34}, {24}, {15}, {7}, {0}, {0}, {0}, {0}},
		{{38}, {27}, {17}, {8}, {0}, {0}, {0}, {0}},
		{{42}, {30}, {19}, {9}, {0}, {0}, {0}, {0}} }
	);

	for (int j = 0; j < batch_size; ++j) {
		for (int k = 0; k < memory_size; ++k) {
			for (int i = 0; i < output_nodes.size(); ++i) {
				const std::string node_name = output_nodes[i];
				//std::cout << "Node: " << node_name << "; Batch: " << j << "; Memory: " << k << std::endl;
				//std::cout << "Calc Output: " << model_getModelResults.getNodesMap().at(node_name)->getOutput()(j, k) << ", Expected Output: " << output(j, k, i) << std::endl;
				assert(model_getModelResults.getNodesMap().at(node_name)->getOutput()(j, k) == output(j, k, i));
			}
		}
	}

	// test values of model error
	Eigen::Tensor<float, 2> model_error(batch_size, memory_size);
	model_error.setValues({
		{242,98,32,2,0,0,0,0},
		{312.5f,144.5f,40.5f,4.5f,0,0,0,0},
		{420.5f,180.5f,60.5f,4.5f,0,0,0,0},
		{512,242,72,8,0,0,0,0},
		{648,288,98,8,0,0,0,0} });
	for (int j = 0; j < batch_size; ++j) {
		for (int k = 0; k < memory_size; ++k) {
			//std::cout << "Batch: " << j << "; Memory: " << k << std::endl;
			//std::cout << "Calc Model Error: " << model_getModelResults.getError()(j, k) << ", Expected Error: " << model_error(j, k) << std::endl;
			assert(model_getModelResults.getError()(j, k) == model_error(j, k));
		}
	}

	// test values of weights
	std::vector<std::string> weight_ids = { "0", "1", "2", "3", "4" };
	Eigen::Tensor<float, 1> weights(weight_ids.size());
	weights.setValues({ -19.624f, -15.744f, -34.572f, 1.0f, 1.0f });
	for (int i = 0; i < weight_ids.size(); ++i) {
		assert(model_getModelResults.getWeightsMap().at(weight_ids[i])->getWeight() == weights(i));
	}
}

int main(int argc, char** argv)
{
	test_allocateForwardPropogationLayerTensors();
	test_getForwardPropogationOperations();
	test_allocateModelErrorTensor();
	test_mapValuesToLayers();
	test_executeForwardPropogationOperations();
	test_executeModelErrorOperations();
	test_executeBackwardPropogationOperations();
	test_executeWeightErrorOperations();
	test_executeWeightUpdateOperations();
	return 0;
}
#endif